#include "hip/hip_runtime.h"
#include "common.cuh"
#include "fattn-common.cuh"
#include "fattn-vec-f16.cuh"

template<int D, int ncols, int parallel_blocks, vec_dot_KQ_f16_t vec_dot_KQ, bool Q_q8_1, dequantize_1_f16_t dequantize_1_v> // D == head size
#if !(defined(GGML_USE_HIPBLAS) && defined(__HIP_PLATFORM_AMD__))
__launch_bounds__(D, 1)
#endif // !(defined(GGML_USE_HIPBLAS) && defined(__HIP_PLATFORM_AMD__))
static __global__ void flash_attn_vec_ext_f16(
        const char * __restrict__ Q,
        const char * __restrict__ K,
        const char * __restrict__ V,
        const char * __restrict__ mask,
        float      * __restrict__ dst,
        float2     * __restrict__ dst_meta,
        const float scale,
        const float max_bias,
        const float m0,
        const float m1,
        const uint32_t n_head_log2,
        const int ne00,
        const int ne01,
        const int ne02,
        const int ne03,
        const int ne10,
        const int ne11,
        const int ne12,
        const int ne13,
        const int ne31,
        const int nb31,
        const int nb01,
        const int nb02,
        const int nb03,
        const int nb11,
        const int nb12,
        const int nb13,
        const int nb21,
        const int nb22,
        const int nb23,
        const int ne0,
        const int ne1,
        const int ne2,
        const int ne3) {
#if FP16_AVAILABLE
    //In this kernel Q, K, V are matrices while i, j, k are matrix indices.

    const int ic0 = (blockIdx.x / parallel_blocks) * ncols; // Index of the Q/QKV column to work on.
    const int ip  =  blockIdx.x % parallel_blocks; // Index in group of blocks running for the same column in parallel.

    const int gqa_ratio = ne02 / ne12; // With grouped query attention there are > 1 Q matrices per K, V matrix.
    const float  * Q_f   = (const float  *) (Q    + nb02* blockIdx.y              + nb01*ic0);
    const float2 * Q_f2  = (const float2 *)  Q_f;
    const char   * K_c   = (const char   *) (K    + nb12*(blockIdx.y / gqa_ratio));
    const char   * V_c   = (const char   *) (V    + nb22*(blockIdx.y / gqa_ratio)); // K and V have same shape
    const half   * maskh = (const half   *)  mask + ne11*ic0;

    const float slopef = get_alibi_slope(max_bias, blockIdx.y, n_head_log2, m0, m1);
    const half  slopeh = __float2half(slopef);

    static_assert(D % (2*WARP_SIZE) == 0, "D not divisible by 2*WARP_SIZE == 64.");
    constexpr int nwarps = D / WARP_SIZE;
    const int tid = WARP_SIZE*threadIdx.y + threadIdx.x;
    __builtin_assume(tid < D);

    __shared__ half KQ[ncols*D];
    half2 * KQ2 = (half2 *) KQ;

    half kqmax[ncols];
#pragma unroll
    for (int j = 0; j < ncols; ++j) {
        kqmax[j] = -HALF_MAX_HALF;
    }
    half kqsum[ncols] = {0.0f};

    __shared__ half kqmax_shared[ncols][WARP_SIZE];
    __shared__ half kqsum_shared[ncols][WARP_SIZE];
#pragma unroll
    for (int j = 0; j < ncols; ++j) {
        if (threadIdx.y == 0) {
            kqmax_shared[j][threadIdx.x] = -HALF_MAX_HALF;
            kqsum_shared[j][threadIdx.x] = 0.0f;
        }
    }
    __syncthreads();

    // Convert Q to half2 and store in registers:
    half2 Q_h2[ncols][D/(2*WARP_SIZE)];
    int   Q_i8[ncols][D/(sizeof(int)*QK8_1) == 0 ? 1 : D/(sizeof(int)*QK8_1)];
    half2 Q_ds[ncols][D/QK8_1 == 0 ? 1 : D/QK8_1];
    if (Q_q8_1) {
#pragma unroll
        for (int j0 = 0; j0 < ncols; j0 += nwarps) {
            const int j = j0 + threadIdx.y;

            int   * tmp_q_i8 = (int   *) &KQ[j*D];
            half2 * tmp_q_ds = (half2 *) (tmp_q_i8 + D/sizeof(int));

            if (ncols > 2 && ic0 + j >= ne01) {
#pragma unroll
                for (int i0 = 0; i0 < D/sizeof(int); i0 += WARP_SIZE) {
                    const int i = i0 + threadIdx.x;

                    tmp_q_i8[i] = 0;
                }
                if (threadIdx.x < D/QK8_1) {
                    tmp_q_ds[threadIdx.x] = make_half2(0.0f, 0.0f);
                }
                continue;
            }

#pragma unroll
            for (int i0 = 0; i0 < D/sizeof(int); i0 += WARP_SIZE) {
                quantize_q8_1_to_shared<half2>(Q_f + j*(nb01/sizeof(float)) + 4*i0, scale, tmp_q_i8, tmp_q_ds);
            }
        }

        __syncthreads();

#pragma unroll
        for (int j = 0; j < ncols; ++j) {
            int   * tmp_q_i8 = (int   *) &KQ[j*D];
            half2 * tmp_q_ds = (half2 *) (tmp_q_i8 + D/sizeof(int));

#pragma unroll
            for (int i0 = 0; i0 < D/sizeof(int); i0 += WARP_SIZE) {
                const int i = i0 + threadIdx.x;

                Q_i8[j][i0/WARP_SIZE] = tmp_q_i8[i];
                Q_ds[j][i0/WARP_SIZE] = tmp_q_ds[i/QI8_1];
            }
        }

        __syncthreads();
    } else {
#pragma unroll
        for (int j = 0; j < ncols; ++j) {
#pragma unroll
            for (int i0 = 0; i0 < D/2; i0 += WARP_SIZE) {
                const int i = i0 + threadIdx.x;

                const float2 tmp = ncols <= 2 || ic0 + j < ne01 ? Q_f2[j*(nb01/sizeof(float2)) + i] : make_float2(0.0f, 0.0f);
                Q_h2[j][i0/WARP_SIZE] = make_half2(scale, scale) * make_half2(tmp.x, tmp.y);
            }
        }
    }


#pragma unroll
    for (int j = 0; j < ncols; ++j) {
        KQ[j*D + tid] = -HALF_MAX_HALF;
    }

    half2 VKQ[ncols] = {{0.0f, 0.0f}};

    const int k_start = parallel_blocks == 1 ? 0 : ip*D;
    for (int k_VKQ_0 = k_start; k_VKQ_0 < ne11; k_VKQ_0 += parallel_blocks*D) {
        // Calculate KQ tile and keep track of new maximum KQ values:

        // For unknown reasons using a half array of size 1 for kqmax_new causes a performance regression,
        // see https://github.com/ggerganov/llama.cpp/pull/7061 .
        // Therefore this variable is defined twice but only used once (so that the compiler can optimize out the unused variable).
        half kqmax_new = kqmax[0];
        half kqmax_new_arr[ncols];
#pragma unroll
        for (int j = 0; j < ncols; ++j) {
            kqmax_new_arr[j] = kqmax[j];
        }

#pragma unroll
        for (int i_KQ_0 = 0; i_KQ_0 < D; i_KQ_0 += nwarps) {
            const int i_KQ = i_KQ_0 + threadIdx.y;

            if ((i_KQ_0 + nwarps > D && i_KQ >= D) || (FATTN_KQ_STRIDE % D != 0 && k_VKQ_0 + i_KQ >= ne11)) {
                break;
            }

#pragma unroll
            for (int j = 0; j < ncols; ++j) {
                half sum = vec_dot_KQ(K_c + (k_VKQ_0 + i_KQ)*nb11, Q_h2[j], Q_i8[j], Q_ds[j]);
                sum = warp_reduce_sum(sum);
                sum += mask ? slopeh*maskh[j*ne11 + k_VKQ_0 + i_KQ] : __float2half(0.0f);

                if (ncols == 1) {
                    kqmax_new        = ggml_cuda_hmax(kqmax_new,        sum);
                } else {
                    kqmax_new_arr[j] = ggml_cuda_hmax(kqmax_new_arr[j], sum);
                }

                if (threadIdx.x == 0) {
                    KQ[j*D + i_KQ] = sum;
                }
            }
        }

#pragma unroll
        for (int j = 0; j < ncols; ++j) {
            half kqmax_new_j = ncols == 1 ? kqmax_new : kqmax_new_arr[j];

            kqmax_new_j = warp_reduce_max(kqmax_new_j);
            if (threadIdx.x == 0) {
                kqmax_shared[j][threadIdx.y] = kqmax_new_j;
            }
        }

        __syncthreads();

#pragma unroll
        for (int j = 0; j < ncols; ++j) {
            half kqmax_new_j = kqmax_shared[j][threadIdx.x];
            kqmax_new_j = warp_reduce_max(kqmax_new_j);

            const half KQ_max_scale = hexp(kqmax[j] - kqmax_new_j);
            kqmax[j] = kqmax_new_j;

            const half val = hexp(KQ[j*D + tid] - kqmax[j]);
            kqsum[j] = kqsum[j]*KQ_max_scale + val;
            KQ[j*D + tid] = val;

            VKQ[j] *= __half2half2(KQ_max_scale);
        }

        __syncthreads();

#pragma unroll
        for (int k0 = 0; k0 < D; k0 += 2) {
            if (FATTN_KQ_STRIDE % D != 0 && k_VKQ_0 + k0 >= ne11) {
                break;
            }

            half2 V_k;
            reinterpret_cast<half&>(V_k.x) = dequantize_1_v(V_c + (k_VKQ_0 + k0 + 0)*nb21, tid);
            reinterpret_cast<half&>(V_k.y) = dequantize_1_v(V_c + (k_VKQ_0 + k0 + 1)*nb21, tid);
#pragma unroll
            for (int j = 0; j < ncols; ++j) {
                VKQ[j] += V_k*KQ2[j*(D/2) + k0/2];
            }
        }

        __syncthreads();
    }

#pragma unroll
    for (int j = 0; j < ncols; ++j) {
        kqsum[j] = warp_reduce_sum(kqsum[j]);
        if (threadIdx.x == 0) {
            kqsum_shared[j][threadIdx.y] = kqsum[j];
        }
    }

    __syncthreads();

#pragma unroll
    for (int j_VKQ = 0; j_VKQ < ncols; ++j_VKQ) {
        if (ncols > 2 && ic0 + j_VKQ >= ne01) {
            break;
        }

        kqsum[j_VKQ] = kqsum_shared[j_VKQ][threadIdx.x];
        kqsum[j_VKQ] = warp_reduce_sum(kqsum[j_VKQ]);

        half dst_val = (__low2half(VKQ[j_VKQ]) + __high2half(VKQ[j_VKQ]));
        if (parallel_blocks == 1) {
            dst_val /= kqsum[j_VKQ];
        }
        const int j_dst = (ic0 + j_VKQ)*parallel_blocks + ip;
        dst[j_dst*D*gridDim.y + D*blockIdx.y + tid] = dst_val;
    }

    if (parallel_blocks != 1 && tid < ncols && (ncols <= 2 || ic0 + tid < ne01)) {
        dst_meta[(ic0 + tid)*gridDim.y*parallel_blocks + blockIdx.y*parallel_blocks + ip] = make_float2(kqmax[tid], kqsum[tid]);
    }
#else
   NO_DEVICE_CODE;
#endif // FP16_AVAILABLE
}

void ggml_cuda_flash_attn_ext_vec_f16(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    ggml_tensor * KQV = dst;
    ggml_tensor * Q   = dst->src[0];

    const int32_t precision = KQV->op_params[2];
    GGML_ASSERT(precision == GGML_PREC_DEFAULT);

    constexpr int cols_per_block  = 1;
    constexpr int parallel_blocks = 4;
    switch (Q->ne[0]) {
        case  64: {
            constexpr int      D = 64;
            constexpr int nwarps = D/WARP_SIZE;
            fattn_kernel_t fattn_kernel = flash_attn_vec_ext_f16<
                D, cols_per_block, parallel_blocks, vec_dot_fattn_vec_KQ_f16<half, D>, false, dequantize_1_f16<half>>;
            launch_fattn<D, parallel_blocks>(ctx, dst, fattn_kernel, nwarps, cols_per_block);
        } break;
        case 128: {
            constexpr int      D = 128;
            constexpr int nwarps = D/WARP_SIZE;
            fattn_kernel_t fattn_kernel = flash_attn_vec_ext_f16<
                D, cols_per_block, parallel_blocks, vec_dot_fattn_vec_KQ_f16<half, D>, false, dequantize_1_f16<half>>;
            launch_fattn<D, parallel_blocks>(ctx, dst, fattn_kernel, nwarps, cols_per_block);
        } break;
        case 256: {
            constexpr int      D = 256;
            constexpr int nwarps = D/WARP_SIZE;
            fattn_kernel_t fattn_kernel = flash_attn_vec_ext_f16<
                D, cols_per_block, parallel_blocks, vec_dot_fattn_vec_KQ_f16<half, D>, false, dequantize_1_f16<half>>;
            launch_fattn<D, parallel_blocks>(ctx, dst, fattn_kernel, nwarps, cols_per_block);
        } break;
        default:
            GGML_ASSERT(false);
            break;
    }
}

template <int D, int cols_per_block, int parallel_blocks, dequantize_1_f16_t dequantize_1_v>
void launch_fattn_tile_f16_K_type(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    constexpr int nwarps = D/WARP_SIZE;
    const ggml_tensor * K = dst->src[1];

    switch (K->type) {
        case GGML_TYPE_Q4_0: {
            fattn_kernel_t fattn_kernel = flash_attn_vec_ext_f16<
                D, cols_per_block, parallel_blocks, vec_dot_fattn_vec_KQ_q4_0<half, D>, true, dequantize_1_v>;
            launch_fattn<D, parallel_blocks>(ctx, dst, fattn_kernel, nwarps, cols_per_block);
        } break;
        case GGML_TYPE_Q4_1: {
            fattn_kernel_t fattn_kernel = flash_attn_vec_ext_f16<
                D, cols_per_block, parallel_blocks, vec_dot_fattn_vec_KQ_q4_1<half, D>, true, dequantize_1_v>;
            launch_fattn<D, parallel_blocks>(ctx, dst, fattn_kernel, nwarps, cols_per_block);
        } break;
        case GGML_TYPE_Q5_0: {
            fattn_kernel_t fattn_kernel = flash_attn_vec_ext_f16<
                D, cols_per_block, parallel_blocks, vec_dot_fattn_vec_KQ_q5_0<half, D>, true, dequantize_1_v>;
            launch_fattn<D, parallel_blocks>(ctx, dst, fattn_kernel, nwarps, cols_per_block);
        } break;
        case GGML_TYPE_Q5_1: {
            fattn_kernel_t fattn_kernel = flash_attn_vec_ext_f16<
                D, cols_per_block, parallel_blocks, vec_dot_fattn_vec_KQ_q5_1<half, D>, true, dequantize_1_v>;
            launch_fattn<D, parallel_blocks>(ctx, dst, fattn_kernel, nwarps, cols_per_block);
        } break;
        case GGML_TYPE_Q8_0: {
            fattn_kernel_t fattn_kernel = flash_attn_vec_ext_f16<
                D, cols_per_block, parallel_blocks, vec_dot_fattn_vec_KQ_q8_0<half, D>, true, dequantize_1_v>;
            launch_fattn<D, parallel_blocks>(ctx, dst, fattn_kernel, nwarps, cols_per_block);
        } break;
        case GGML_TYPE_F16: {
            fattn_kernel_t fattn_kernel = flash_attn_vec_ext_f16<
                D, cols_per_block, parallel_blocks, vec_dot_fattn_vec_KQ_f16<half, D>, false, dequantize_1_v>;
            launch_fattn<D, parallel_blocks>(ctx, dst, fattn_kernel, nwarps, cols_per_block);
        } break;
        default:
            GGML_ASSERT(false);
            break;
    }
}

template <int cols_per_block, int parallel_blocks, dequantize_1_f16_t dequantize_1_v>
void launch_fattn_vec_f16_64_128(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * Q = dst->src[0];
    switch (Q->ne[0]) {
        case  64: {
            GGML_ASSERT(Q->type == GGML_TYPE_F16 && "Quantized K cache not supported for head size 64.");
            constexpr int D      = 64;
            constexpr int nwarps = D/WARP_SIZE;
            fattn_kernel_t fattn_kernel = flash_attn_vec_ext_f16<
                D, cols_per_block, parallel_blocks, vec_dot_fattn_vec_KQ_f16<half, D>, false, dequantize_1_v>;
            launch_fattn<D, parallel_blocks>(ctx, dst, fattn_kernel, nwarps, cols_per_block);
        } break;
        case 128: {
            constexpr int D      = 128;
            launch_fattn_tile_f16_K_type<D, cols_per_block, parallel_blocks, dequantize_1_v>(ctx, dst);
        } break;
        default: {
            GGML_ASSERT(false && "FlashAttention without tensor cores only supports head sizes 64 and 128.");
        } break;
    }
}

template <int cols_per_block, int parallel_blocks>
void launch_fattn_vec_f16_V_type(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * V = dst->src[2];

    switch (V->type) {
        case GGML_TYPE_Q4_0:
            launch_fattn_vec_f16_64_128<cols_per_block, parallel_blocks, dequantize_1_q4_0<half>>(ctx, dst);
            break;
        case GGML_TYPE_Q4_1:
            launch_fattn_vec_f16_64_128<cols_per_block, parallel_blocks, dequantize_1_q4_1<half>>(ctx, dst);
            break;
        case GGML_TYPE_Q5_0:
            launch_fattn_vec_f16_64_128<cols_per_block, parallel_blocks, dequantize_1_q5_0<half>>(ctx, dst);
            break;
        case GGML_TYPE_Q5_1:
            launch_fattn_vec_f16_64_128<cols_per_block, parallel_blocks, dequantize_1_q5_1<half>>(ctx, dst);
            break;
        case GGML_TYPE_Q8_0:
            launch_fattn_vec_f16_64_128<cols_per_block, parallel_blocks, dequantize_1_q8_0<half>>(ctx, dst);
            break;
        case GGML_TYPE_F16:
            launch_fattn_vec_f16_64_128<cols_per_block, parallel_blocks, dequantize_1_f16<half>>(ctx, dst);
            break;
        default:
            GGML_ASSERT(false);
            break;
    }
}

void ggml_cuda_flash_attn_ext_vec_f16_no_mma(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * KQV = dst;
    const ggml_tensor * Q   = dst->src[0];

    const int32_t precision = KQV->op_params[2];
    GGML_ASSERT(precision == GGML_PREC_DEFAULT);

    if (Q->ne[1] == 1) {
        constexpr int cols_per_block  = 1;
        constexpr int parallel_blocks = 4;
        launch_fattn_vec_f16_V_type<cols_per_block, parallel_blocks>(ctx, dst);
        return;
    }

    if (Q->ne[1] == 2) {
        constexpr int cols_per_block  = 2;
        constexpr int parallel_blocks = 4;
        launch_fattn_vec_f16_V_type<cols_per_block, parallel_blocks>(ctx, dst);
        return;
    }

    if (Q->ne[1] <= 4) {
        constexpr int cols_per_block  = 4;
        constexpr int parallel_blocks = 4;
        launch_fattn_vec_f16_V_type<cols_per_block, parallel_blocks>(ctx, dst);
        return;
    }

    if (Q->ne[1] <= 8) {
        constexpr int cols_per_block  = 8;
        constexpr int parallel_blocks = 4;
        launch_fattn_vec_f16_V_type<cols_per_block, parallel_blocks>(ctx, dst);
        return;
    }

    constexpr int cols_per_block  = 8;
    constexpr int parallel_blocks = 1;
    launch_fattn_vec_f16_V_type<cols_per_block, parallel_blocks>(ctx, dst);
}
