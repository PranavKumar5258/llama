#include "hip/hip_runtime.h"
#include "common.cuh"
#include "fattn-common.cuh"
#include "fattn-vec-f32.cuh"

template<int D, int ncols, int parallel_blocks, vec_dot_KQ_f32_t vec_dot_KQ, bool Q_q8_1, dequantize_1_f32_t dequantize_1_v> // D == head size
#if !(defined(GGML_USE_HIPBLAS) && defined(__HIP_PLATFORM_AMD__))
__launch_bounds__(D, 1)
#endif // !(defined(GGML_USE_HIPBLAS) && defined(__HIP_PLATFORM_AMD__))
static __global__ void flash_attn_vec_ext_f32(
        const char * __restrict__ Q,
        const char * __restrict__ K,
        const char * __restrict__ V,
        const char * __restrict__ mask,
        float      * __restrict__ dst,
        float2     * __restrict__ dst_meta,
        const float scale,
        const float max_bias,
        const float m0,
        const float m1,
        const uint32_t n_head_log2,
        const int ne00,
        const int ne01,
        const int ne02,
        const int ne03,
        const int ne10,
        const int ne11,
        const int ne12,
        const int ne13,
        const int ne31,
        const int nb31,
        const int nb01,
        const int nb02,
        const int nb03,
        const int nb11,
        const int nb12,
        const int nb13,
        const int nb21,
        const int nb22,
        const int nb23,
        const int ne0,
        const int ne1,
        const int ne2,
        const int ne3) {
    //In this kernel Q, K, V are matrices while i, j, k are matrix indices.

    const int ic0 = (blockIdx.x / parallel_blocks) * ncols; // Index of the Q/QKV column to work on.
    const int ip  =  blockIdx.x % parallel_blocks; // Index in group of blocks running for the same column in parallel.

    const int gqa_ratio = ne02 / ne12; // With grouped query attention there are > 1 Q matrices per K, V matrix.
    const float  * Q_f   = (const float  *) (Q    + nb02* blockIdx.y              + nb01*ic0);
    const float2 * Q_f2  = (const float2 *)  Q_f;
    const char   * K_c   = (const char   *) (K    + nb12*(blockIdx.y / gqa_ratio));
    const char   * V_c   = (const char   *) (V    + nb22*(blockIdx.y / gqa_ratio)); // K and V have same shape
    const half   * maskh = (const half   *)  mask + ne11*ic0;

    const float slope = get_alibi_slope(max_bias, blockIdx.y, n_head_log2, m0, m1);

    static_assert(D % (2*WARP_SIZE) == 0, "D not divisible by 2*WARP_SIZE == 64.");
    constexpr int nwarps = D / WARP_SIZE;
    const int tid = WARP_SIZE*threadIdx.y + threadIdx.x;
    __builtin_assume(tid < D);

    __shared__ float KQ[ncols*D];
#pragma unroll
    for (int j = 0; j < ncols; ++j) {
        KQ[j*D + tid] = -FLT_MAX/2.0f;
    }

    float kqmax[ncols];
#pragma unroll
    for (int j = 0; j < ncols; ++j) {
        kqmax[j] = -FLT_MAX/2.0f;
    }
    float kqsum[ncols] = {0.0f};

    __shared__ float kqmax_shared[ncols][WARP_SIZE];
    __shared__ float kqsum_shared[ncols][WARP_SIZE];
#pragma unroll
    for (int j = 0; j < ncols; ++j) {
        if (threadIdx.y == 0) {
            kqmax_shared[j][threadIdx.x] = -FLT_MAX/2.0f;
            kqsum_shared[j][threadIdx.x] = 0.0f;
        }
    }
    __syncthreads();

    // Convert Q to half2 and store in registers:
    float2 Q_h2[ncols][D/(2*WARP_SIZE)];
    int    Q_i8[ncols][D/(sizeof(int)*QK8_1) == 0 ? 1 : D >= D/(sizeof(int)*QK8_1)];
    float2 Q_ds[ncols][D/QK8_1 == 0 ? 1 : D/QK8_1];
    if (Q_q8_1) {
#pragma unroll
        for (int j0 = 0; j0 < ncols; j0 += nwarps) {
            const int j = j0 + threadIdx.y;

            int    * tmp_q_i8 = (int    *) &KQ[j*D];
            float2 * tmp_q_ds = (float2 *) (tmp_q_i8 + D/sizeof(int));

            if (ncols > 2 && ic0 + j >= ne01) {
#pragma unroll
                for (int i0 = 0; i0 < D/sizeof(int); i0 += WARP_SIZE) {
                    const int i = i0 + threadIdx.x;

                    tmp_q_i8[i] = 0;
                }
                if (threadIdx.x < D/QK8_1) {
                    tmp_q_ds[threadIdx.x] = make_float2(0.0f, 0.0f);
                }
                continue;
            }

#pragma unroll
            for (int i0 = 0; i0 < D/sizeof(int); i0 += WARP_SIZE) {
                quantize_q8_1_to_shared<float2>(Q_f + j*(nb01/sizeof(float)) + 4*i0, scale, tmp_q_i8, tmp_q_ds);
            }
        }

        __syncthreads();

#pragma unroll
        for (int j = 0; j < ncols; ++j) {
            int    * tmp_q_i8 = (int    *) &KQ[j*D];
            float2 * tmp_q_ds = (float2 *) (tmp_q_i8 + D/sizeof(int));

#pragma unroll
            for (int i0 = 0; i0 < D/sizeof(int); i0 += WARP_SIZE) {
                const int i = i0 + threadIdx.x;

                Q_i8[j][i0/WARP_SIZE] = tmp_q_i8[i];
                Q_ds[j][i0/WARP_SIZE] = tmp_q_ds[i/QI8_1];
            }
        }

        __syncthreads();
    } else {
#pragma unroll
        for (int j = 0; j < ncols; ++j) {
#pragma unroll
            for (int i0 = 0; i0 < D/2; i0 += WARP_SIZE) {
                const int i = i0 + threadIdx.x;

                Q_h2[j][i0/WARP_SIZE]    = ncols <= 2 || ic0 + j ? Q_f2[j*(nb01/sizeof(float2)) + i] : make_float2(0.0f, 0.0f);
                Q_h2[j][i0/WARP_SIZE].x *= scale;
                Q_h2[j][i0/WARP_SIZE].y *= scale;
            }
        }
    }

    float VKQ[ncols] = {0.0f};

    const int k_start = parallel_blocks == 1 ? 0 : ip*D;
    for (int k_VKQ_0 = k_start; k_VKQ_0 < ne11; k_VKQ_0 += parallel_blocks*D) {
        // Calculate KQ tile and keep track of new maximum KQ values:

        float kqmax_new_arr[ncols];
#pragma unroll
        for (int j = 0; j < ncols; ++j) {
            kqmax_new_arr[j] = kqmax[j];
        }

#pragma unroll
        for (int i_KQ_0 = 0; i_KQ_0 < D; i_KQ_0 += nwarps) {
            const int i_KQ = i_KQ_0 + threadIdx.y;

            if ((i_KQ_0 + nwarps > D && i_KQ >= D) || (FATTN_KQ_STRIDE % D != 0 && k_VKQ_0 + i_KQ >= ne11)) {
                break;
            }

#pragma unroll
            for (int j = 0; j < ncols; ++j) {
                float sum = vec_dot_KQ(K_c + (k_VKQ_0 + i_KQ)*nb11, Q_h2[j], Q_i8[j], Q_ds[j]);
                sum = warp_reduce_sum(sum);
                sum += mask ? slope*__half2float(maskh[j*ne11 + k_VKQ_0 + i_KQ]) : 0.0f;

                kqmax_new_arr[j] = fmaxf(kqmax_new_arr[j], sum);

                if (threadIdx.x == 0) {
                    KQ[j*D + i_KQ] = sum;
                }
            }
        }

#pragma unroll
        for (int j = 0; j < ncols; ++j) {
            float kqmax_new_j = kqmax_new_arr[j];

            kqmax_new_j = warp_reduce_max(kqmax_new_j);
            if (threadIdx.x == 0) {
                kqmax_shared[j][threadIdx.y] = kqmax_new_j;
            }
        }

        __syncthreads();

#pragma unroll
        for (int j = 0; j < ncols; ++j) {
            float kqmax_new_j = kqmax_shared[j][threadIdx.x];
            kqmax_new_j = warp_reduce_max(kqmax_new_j);

            const float KQ_max_scale = expf(kqmax[j] - kqmax_new_j);
            kqmax[j] = kqmax_new_j;

            const float val = expf(KQ[j*D + tid] - kqmax[j]);
            kqsum[j] = kqsum[j]*KQ_max_scale + val;
            KQ[j*D + tid] = val;

            VKQ[j] *= KQ_max_scale;
        }

        __syncthreads();

#pragma unroll
        for (int k = 0; k < D; ++k) {
            if (FATTN_KQ_STRIDE % D != 0 && k_VKQ_0 + k >= ne11) {
                break;
            }

            const float V_ki = dequantize_1_v(V_c + (k_VKQ_0 + k)*nb21, tid);
#pragma unroll
            for (int j = 0; j < ncols; ++j) {
                VKQ[j] += V_ki*KQ[j*D + k];
            }
        }

        __syncthreads();
    }

#pragma unroll
    for (int j = 0; j < ncols; ++j) {
        kqsum[j] = warp_reduce_sum(kqsum[j]);
        if (threadIdx.x == 0) {
            kqsum_shared[j][threadIdx.y] = kqsum[j];
        }
    }

    __syncthreads();

#pragma unroll
    for (int j_VKQ = 0; j_VKQ < ncols; ++j_VKQ) {
        if (ncols > 2 && ic0 + j_VKQ >= ne01) {
            break;
        }

        kqsum[j_VKQ] = kqsum_shared[j_VKQ][threadIdx.x];
        kqsum[j_VKQ] = warp_reduce_sum(kqsum[j_VKQ]);

        float dst_val = VKQ[j_VKQ];
        if (parallel_blocks == 1) {
            dst_val /= kqsum[j_VKQ];
        }
        const int j_dst = (ic0 + j_VKQ)*parallel_blocks + ip;
        dst[j_dst*D*gridDim.y + D*blockIdx.y + tid] = dst_val;
    }

    if (parallel_blocks != 1 && tid < ncols && (ncols <= 2 || ic0 + tid < ne01)) {
        dst_meta[(ic0 + tid)*gridDim.y*parallel_blocks + blockIdx.y*parallel_blocks + ip] = make_float2(kqmax[tid], kqsum[tid]);
    }
}

template <int D, int cols_per_block, int parallel_blocks, dequantize_1_f32_t dequantize_1_v>
void launch_fattn_tile_f32_K_type(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    constexpr int nwarps = D/WARP_SIZE;
    const ggml_tensor * K = dst->src[1];

    switch (K->type) {
        case GGML_TYPE_Q4_0: {
            fattn_kernel_t fattn_kernel = flash_attn_vec_ext_f32<D, cols_per_block, parallel_blocks, vec_dot_fattn_vec_KQ_q4_0<float, D>, true, dequantize_1_v>;
            launch_fattn<D, parallel_blocks>(ctx, dst, fattn_kernel, nwarps, cols_per_block);
        } break;
        case GGML_TYPE_Q4_1: {
            fattn_kernel_t fattn_kernel = flash_attn_vec_ext_f32<D, cols_per_block, parallel_blocks, vec_dot_fattn_vec_KQ_q4_1<float, D>, true, dequantize_1_v>;
            launch_fattn<D, parallel_blocks>(ctx, dst, fattn_kernel, nwarps, cols_per_block);
        } break;
        case GGML_TYPE_Q5_0: {
            fattn_kernel_t fattn_kernel = flash_attn_vec_ext_f32<D, cols_per_block, parallel_blocks, vec_dot_fattn_vec_KQ_q5_0<float, D>, true, dequantize_1_v>;
            launch_fattn<D, parallel_blocks>(ctx, dst, fattn_kernel, nwarps, cols_per_block);
        } break;
        case GGML_TYPE_Q5_1: {
            fattn_kernel_t fattn_kernel = flash_attn_vec_ext_f32<D, cols_per_block, parallel_blocks, vec_dot_fattn_vec_KQ_q5_1<float, D>, true, dequantize_1_v>;
            launch_fattn<D, parallel_blocks>(ctx, dst, fattn_kernel, nwarps, cols_per_block);
        } break;
        case GGML_TYPE_Q8_0: {
            fattn_kernel_t fattn_kernel = flash_attn_vec_ext_f32<D, cols_per_block, parallel_blocks, vec_dot_fattn_vec_KQ_q8_0<float, D>, true, dequantize_1_v>;
            launch_fattn<D, parallel_blocks>(ctx, dst, fattn_kernel, nwarps, cols_per_block);
        } break;
        case GGML_TYPE_F16: {
            fattn_kernel_t fattn_kernel = flash_attn_vec_ext_f32<D, cols_per_block, parallel_blocks, vec_dot_fattn_vec_KQ_f16<float, D>, false, dequantize_1_v>;
            launch_fattn<D, parallel_blocks>(ctx, dst, fattn_kernel, nwarps, cols_per_block);
        } break;
        default:
            GGML_ASSERT(false);
            break;
    }
}

template <int cols_per_block, int parallel_blocks, dequantize_1_f32_t dequantize_1_v>
void launch_fattn_vec_f32_64_128(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * Q = dst->src[0];
    switch (Q->ne[0]) {
        case  64: {
            GGML_ASSERT(Q->type == GGML_TYPE_F16 && "Quantized K cache not supported for head size 64.");
            constexpr int D      = 64;
            constexpr int nwarps = D/WARP_SIZE;
            fattn_kernel_t fattn_kernel = flash_attn_vec_ext_f32<
                D, cols_per_block, parallel_blocks, vec_dot_fattn_vec_KQ_f16<float, D>, false, dequantize_1_v>;
            launch_fattn<D, parallel_blocks>(ctx, dst, fattn_kernel, nwarps, cols_per_block);
        } break;
        case 128: {
            constexpr int D      = 128;
            launch_fattn_tile_f32_K_type<D, cols_per_block, parallel_blocks, dequantize_1_v>(ctx, dst);
        } break;
        default: {
            GGML_ASSERT(false && "FlashAttention without tensor cores only supports head sizes 64 and 128.");
        } break;
    }
}

template <int cols_per_block, int parallel_blocks>
void launch_fattn_vec_f32_V_type(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * V = dst->src[2];

    switch (V->type) {
        case GGML_TYPE_Q4_0:
            launch_fattn_vec_f32_64_128<cols_per_block, parallel_blocks, dequantize_1_q4_0<float>>(ctx, dst);
            break;
        case GGML_TYPE_Q4_1:
            launch_fattn_vec_f32_64_128<cols_per_block, parallel_blocks, dequantize_1_q4_1<float>>(ctx, dst);
            break;
        case GGML_TYPE_Q5_0:
            launch_fattn_vec_f32_64_128<cols_per_block, parallel_blocks, dequantize_1_q5_0<float>>(ctx, dst);
            break;
        case GGML_TYPE_Q5_1:
            launch_fattn_vec_f32_64_128<cols_per_block, parallel_blocks, dequantize_1_q5_1<float>>(ctx, dst);
            break;
        case GGML_TYPE_Q8_0:
            launch_fattn_vec_f32_64_128<cols_per_block, parallel_blocks, dequantize_1_q8_0<float>>(ctx, dst);
            break;
        case GGML_TYPE_F16:
            launch_fattn_vec_f32_64_128<cols_per_block, parallel_blocks, dequantize_1_f16<float>>(ctx, dst);
            break;
        default:
            GGML_ASSERT(false);
            break;
    }
}

void ggml_cuda_flash_attn_ext_vec_f32(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * Q = dst->src[0];

    if (Q->ne[1] == 1) {
        constexpr int cols_per_block  = 1;
        constexpr int parallel_blocks = 4;
        launch_fattn_vec_f32_V_type<cols_per_block, parallel_blocks>(ctx, dst);
        return;
    }

    if (Q->ne[1] == 2) {
        constexpr int cols_per_block  = 2;
        constexpr int parallel_blocks = 4;
        launch_fattn_vec_f32_V_type<cols_per_block, parallel_blocks>(ctx, dst);
        return;
    }

    if (Q->ne[1] <= 4) {
        constexpr int cols_per_block  = 4;
        constexpr int parallel_blocks = 4;
        launch_fattn_vec_f32_V_type<cols_per_block, parallel_blocks>(ctx, dst);
        return;
    }

    if (Q->ne[1] <= 8) {
        constexpr int cols_per_block  = 8;
        constexpr int parallel_blocks = 4;
        launch_fattn_vec_f32_V_type<cols_per_block, parallel_blocks>(ctx, dst);
        return;
    }

    constexpr int cols_per_block  = 8;
    constexpr int parallel_blocks = 1;
    launch_fattn_vec_f32_V_type<cols_per_block, parallel_blocks>(ctx, dst);
}
